/*
 * Copyright (c) 2018-2019 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA Corporation is strictly prohibited.
 *
 * Edited by Marcos Luciano
 * https://www.github.com/marcoslucianops
 *
 */

#include <hip/hip_runtime.h>

#include <stdint.h>
#include <stdio.h>
#include <string.h>

inline __device__ float sigmoidGPU(const float& x) { return 1.0f / (1.0f + __expf(-x)); }

__global__ void gpuYoloLayer(const float* input, float* output, const uint gridSizeX, const uint gridSizeY, const uint numOutputClasses,
                               const uint numBBoxes, const float scale_x_y)
{
    uint x_id = blockIdx.x * blockDim.x + threadIdx.x;
    uint y_id = blockIdx.y * blockDim.y + threadIdx.y;
    uint z_id = blockIdx.z * blockDim.z + threadIdx.z;

    if ((x_id >= gridSizeX) || (y_id >= gridSizeY) || (z_id >= numBBoxes))
    {
        return;
    }

    const int numGridCells = gridSizeX * gridSizeY;
    const int bbindex = y_id * gridSizeX + x_id;

    const float alpha = scale_x_y;
    const float beta = -0.5 * (scale_x_y - 1);

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 0)]
        = sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 0)]) * alpha + beta;

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 1)]
        = sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 1)]) * alpha + beta;

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)]
        = __expf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)]);

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)]
        = __expf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)]);

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)]
        = sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)]);

    for (uint i = 0; i < numOutputClasses; ++i)
    {
        output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))]
            = sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))]);
    }
}

hipError_t cudaYoloLayer(const void* input, void* output, const uint& batchSize, const uint& gridSizeX, const uint& gridSizeY,
                            const uint& numOutputClasses, const uint& numBBoxes, uint64_t outputSize, hipStream_t stream,
                            const float modelScale);

hipError_t cudaYoloLayer(const void* input, void* output, const uint& batchSize, const uint& gridSizeX, const uint& gridSizeY,
                            const uint& numOutputClasses, const uint& numBBoxes, uint64_t outputSize, hipStream_t stream,
                            const float modelScale)
{
    dim3 threads_per_block(16, 16, 4);
    dim3 number_of_blocks((gridSizeX / threads_per_block.x) + 1,
                          (gridSizeY / threads_per_block.y) + 1,
                          (numBBoxes / threads_per_block.z) + 1);
    for (unsigned int batch = 0; batch < batchSize; ++batch)
    {
        gpuYoloLayer<<<number_of_blocks, threads_per_block, 0, stream>>>(
            reinterpret_cast<const float*>(input) + (batch * outputSize),
            reinterpret_cast<float*>(output) + (batch * outputSize), gridSizeX, gridSizeY, numOutputClasses,
            numBBoxes, modelScale);
    }
    return hipGetLastError();
}
