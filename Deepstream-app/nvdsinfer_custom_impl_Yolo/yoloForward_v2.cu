/*
 * Created by Marcos Luciano
 * https://www.github.com/marcoslucianops
 */

#include <hip/hip_runtime.h>

#include <stdint.h>
#include <stdio.h>
#include <string.h>

inline __device__ float sigmoidGPU(const float& x) { return 1.0f / (1.0f + __expf(-x)); }

__global__ void gpuRegionLayer(const float* input, float* output, const uint gridSizeX, const uint gridSizeY, const uint numOutputClasses,
                               const uint numBBoxes)
{
    uint x_id = blockIdx.x * blockDim.x + threadIdx.x;
    uint y_id = blockIdx.y * blockDim.y + threadIdx.y;
    uint z_id = blockIdx.z * blockDim.z + threadIdx.z;

    if ((x_id >= gridSizeX) || (y_id >= gridSizeY) || (z_id >= numBBoxes))
    {
        return;
    }

    const int numGridCells = gridSizeX * gridSizeY;
    const int bbindex = y_id * gridSizeX + x_id;

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 0)]
        = sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 0)]);

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 1)]
        = sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 1)]);

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)]
        = __expf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)]);

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)]
        = __expf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)]);

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)]
        = sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)]);

    float temp = 1.0;
    int i;
    float sum = 0;
    float largest = -INFINITY;
    for(i = 0; i < numOutputClasses; ++i){
        int val = input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))];
        largest = (val>largest) ? val : largest;
    }
    for(i = 0; i < numOutputClasses; ++i){
        float e = exp(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))] / temp - largest / temp);
        sum += e;
        output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))] = e;
    }
    for(i = 0; i < numOutputClasses; ++i){
        output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))] /= sum;
    }
}

hipError_t cudaYoloLayer_v2(const void* input, void* output, const uint& batchSize, const uint& gridSizeX, const uint& gridSizeY,
                            const uint& numOutputClasses, const uint& numBBoxes, uint64_t outputSize, hipStream_t stream);

hipError_t cudaYoloLayer_v2(const void* input, void* output, const uint& batchSize, const uint& gridSizeX, const uint& gridSizeY,
                            const uint& numOutputClasses, const uint& numBBoxes, uint64_t outputSize, hipStream_t stream)
{
    dim3 threads_per_block(16, 16, 4);
    dim3 number_of_blocks((gridSizeX / threads_per_block.x) + 1,
                          (gridSizeY / threads_per_block.y) + 1,
                          (numBBoxes / threads_per_block.z) + 1);
    for (unsigned int batch = 0; batch < batchSize; ++batch)
    {
        gpuRegionLayer<<<number_of_blocks, threads_per_block, 0, stream>>>(
            reinterpret_cast<const float*>(input) + (batch * outputSize),
            reinterpret_cast<float*>(output) + (batch * outputSize), gridSizeX, gridSizeY, numOutputClasses,
            numBBoxes);
    }
    return hipGetLastError();
}
