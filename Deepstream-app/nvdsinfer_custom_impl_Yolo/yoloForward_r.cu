/*
 * Created by Marcos Luciano
 * https://www.github.com/marcoslucianops
 */

#include <hip/hip_runtime.h>

#include <stdint.h>
#include <stdio.h>
#include <string.h>

inline __device__ float sigmoidGPU(const float& x) { return 1.0f / (1.0f + __expf(-x)); }

__global__ void gpuYoloLayer_r(const float* input, float* output, const uint gridSizeX, const uint gridSizeY, const uint numOutputClasses,
                               const uint numBBoxes, const float scale_x_y)
{
    uint x_id = blockIdx.x * blockDim.x + threadIdx.x;
    uint y_id = blockIdx.y * blockDim.y + threadIdx.y;
    uint z_id = blockIdx.z * blockDim.z + threadIdx.z;

    if ((x_id >= gridSizeX) || (y_id >= gridSizeY) || (z_id >= numBBoxes))
    {
        return;
    }

    const int numGridCells = gridSizeX * gridSizeY;
    const int bbindex = y_id * gridSizeX + x_id;

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 0)]
        = sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 0)]) * 2.0 - 0.5;

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 1)]
        = sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 1)]) * 2.0 - 0.5;

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)]
        = pow(sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)]) * 2, 2);

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)]
        = pow(sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)]) * 2, 2);

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)]
        = sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)]);

    for (uint i = 0; i < numOutputClasses; ++i)
    {
        output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))]
            = sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))]);
    }
}

hipError_t cudaYoloLayer_r(const void* input, void* output, const uint& batchSize, const uint& gridSizeX, const uint& gridSizeY,
                            const uint& numOutputClasses, const uint& numBBoxes, uint64_t outputSize, hipStream_t stream,
                            const float modelScale);

hipError_t cudaYoloLayer_r(const void* input, void* output, const uint& batchSize, const uint& gridSizeX, const uint& gridSizeY,
                            const uint& numOutputClasses, const uint& numBBoxes, uint64_t outputSize, hipStream_t stream,
                            const float modelScale)
{
    dim3 threads_per_block(16, 16, 4);
    dim3 number_of_blocks((gridSizeX / threads_per_block.x) + 1,
                          (gridSizeY / threads_per_block.y) + 1,
                          (numBBoxes / threads_per_block.z) + 1);
    for (unsigned int batch = 0; batch < batchSize; ++batch)
    {
        gpuYoloLayer_r<<<number_of_blocks, threads_per_block, 0, stream>>>(
            reinterpret_cast<const float*>(input) + (batch * outputSize),
            reinterpret_cast<float*>(output) + (batch * outputSize), gridSizeX, gridSizeY, numOutputClasses,
            numBBoxes, modelScale);
    }
    return hipGetLastError();
}
